#include <stdio.h>

int main () {
	int  deviceCount;
	hipDeviceProp_t devProp;

	hipGetDeviceCount (&deviceCount);

	printf("Found %d deviced\n", deviceCount);

	for (int device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&devProp, device);

		printf("Device %d\n", device);
		printf("Compute capability      : %d.%d\n", devProp.major, devProp.minor);
		printf("Name                    : %s\n", devProp.name);
		printf("Total Global Memory     : %lu\n", devProp.totalGlobalMem);
		printf("Shared Memory per block : %lu\n", devProp.sharedMemPerBlock);
		printf("Registers per block     : %d\n", devProp.regsPerBlock);
		printf("Warp size               : %d\n", devProp.warpSize);
		printf("Max Threads per block   : %d\n", devProp.maxThreadsPerBlock);
		printf("Total constant memory   : %lu\n", devProp.totalConstMem);
		printf("Clock Rate              : %d\n", devProp.clockRate);
		printf("Texture Alignment       : %lu\n", devProp.textureAlignment);
		printf("Device Overlap          : %d\n", devProp.deviceOverlap);
		printf("Multiprocessor Count    : %d\n", devProp.multiProcessorCount);

		printf("Max Threads Dim         : %d %d %d\n", devProp.maxThreadsDim[0],
						devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);

		printf("Max Grid Size           : %d %d %d\n", devProp.maxGridSize[0],
						devProp.maxGridSize[1], devProp.maxGridSize[2]);
	}

	return 0;
}
